#include "hip/hip_runtime.h"
extern "C" {
#include "freq_cuda.h"
}

__global__ void C3_kernel(size_t n, size_t *A_com, size_t *A_unc, size_t *c3) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    int j, k, l, lb, up, clb, cup, llb;
    lb = A_com[i];
    up = A_com[i + 1];
    for (j = lb; j < up; j++) {
      clb = A_com[A_unc[j]];
      cup = A_com[A_unc[j] + 1];
      llb = lb;
      for (k = clb; k < cup; k++) {
        for (l = llb; l < up; l++) {
          if (A_unc[k] == A_unc[l]) {
            c3[i]++;
            llb = l + 1;
            break;
          } else if (A_unc[k] < A_unc[l]) {
            llb = l;
            break;
          } else {
            llb = l + 1;
          }
        }
      }
    }
    c3[i] /= 2;
  }
}

extern "C" {
void cuda_c3(csx A, freq f) {
  // Allocate device memory
  size_t *A_com, *A_unc, *c3;
  hipMalloc(&A_com, (A->v + 1) * sizeof(size_t));
  hipMalloc(&A_unc, A->e * sizeof(size_t));
  hipMalloc(&c3, A->v * sizeof(size_t));
  hipMemset(c3, 0, A->v * sizeof(size_t));

  // Copy A to device memory
  hipMemcpy(A_com, A->com, (A->v + 1) * sizeof(size_t),
             hipMemcpyHostToDevice);
  hipMemcpy(A_unc, A->unc, A->e * sizeof(size_t), hipMemcpyHostToDevice);

  // Find necessary number of blocks and number of threads
  dim3 dimGrid(1, 1);
  dim3 dimBlock(A->v);
  if (A->v > 1024) {
    size_t grid = (A->v + 1024 - 1) / 1024;
    dimGrid.x = grid;
    dimBlock.x = 1024;
  }

  // Launch kernel
  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, &start);

  C3_kernel<<<dimGrid, dimBlock>>>(A->v, A_com, A_unc, c3);
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &end);
  f->s4_ms = get_elapsed_ms(start, end);

  // Copy B back to host memory
  hipMemcpy(f->s4, c3, A->v * sizeof(size_t), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(A_com);
  hipFree(A_unc);
  hipFree(c3);
}
}
